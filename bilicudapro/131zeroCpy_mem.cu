#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#include "00common.h"

__global__ void zeroCopyMemory(float *input)
{
    printf("GPU zeroCopy memory:%.2f\n", *input);
}

int main(int argc, char** argv)
{
    // get GPU device count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if (error != hipSuccess || nDeviceNumber == 0)
    {
        std::cout << "No CUDA campatable GPU found" << std::endl;
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if (error != hipSuccess)
    {
        std::cout << "Fail to set GPU 0 for computing" << std::endl;
        return -1;
    }
    else 
    {
        std::cout << "Set GPU 0 for computing" << std::endl;
    }

    
    // calculate on gpu
    dim3 block (1);
    dim3 grid (1);

    float *h_zeroCoyMem = nullptr;

    ErrorCheck(hipHostAlloc((float**)&h_zeroCoyMem, sizeof(float)), __FILE__, __LINE__);

    *h_zeroCoyMem = 4.8;

    zeroCopyMemory<<<grid, block>>>(h_zeroCoyMem);
    hipDeviceSynchronize();
    std::cout << "CPU pageLocked memory: " << *h_zeroCoyMem <<std::endl;


    hipFree (h_zeroCoyMem);
    hipDeviceReset();
    return 0;
}