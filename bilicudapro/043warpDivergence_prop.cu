#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "00common.h"

__global__ void mathKernel(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    // 分支特点： warp 分支只会发生在同一 warp 内
    // 如果代码中的条件判断值与线程 id 关联，则以线程束为基本单元访问数
    if ((tid / warpSize) % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }
    
    c[tid] = ia + ib;
}

int main(int argc, char **argv)
{
    // get GPU device count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if (error != hipSuccess || nDeviceNumber == 0)
    {
        std::cout << "No CUDA campatable GPU found" << std::endl;
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if (error != hipSuccess || nDeviceNumber == 0)
    {
        std::cout << "Fail to set GPU 0 for computing" << std::endl;
        return -1;
    }
    else 
    {
        std::cout << "Set GPU 0 for computing" << std::endl;
    }

    // set up data size
    int size = 64;
    int blocksize = 64;

    std::cout << "Data size : " << size << std::endl;

    dim3 block (blocksize, 1);
    dim3 grid ((size + block.x - 1) / block.x);

    float *d_C;
    size_t nBytes = size * sizeof(float);
    error = ErrorCheck(hipMalloc((float**)&d_C, nBytes), __FILE__, __LINE__);
    if (error != hipSuccess)
    {
        std::cout << "Fail to set GPU 0 for computing" << std::endl;
        return -1;
    }

    mathKernel<<<grid, block>>>(d_C);
    hipDeviceSynchronize();

    hipFree(d_C);
    hipDeviceReset();
    return 0;
}