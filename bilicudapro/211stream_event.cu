#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstring>

#include "00common.h"

void initialData(float* ip, int size)
{
    time_t t;
    srand((unsigned) time(&t));
    // std::cout << "Matrix is: ";
    for (int i = 0; i < size; ++i)
    {
        ip[i] = static_cast<float>(rand() & 0xff) / 10.0;
        // std::cout << std::fixed << std::setprecision(2) << ip[i] << " ";
    }
    // std::cout <<std::endl;
    return;
}

__global__ void sumArraysOnGPU(float* A, float* B, float* C, const int N)
{
    int i = threadIdx.x;
    if(i < N)
    {
        C[i] = A[i] + B[i];
    }
}

int main(int argc, char** argv)
{
    // get GPU device count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if (error != hipSuccess || nDeviceNumber == 0)
    {
        std::cout << "No CUDA campatable GPU found" << std::endl;
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if (error != hipSuccess || nDeviceNumber == 0)
    {
        std::cout << "Fail to set GPU 0 for computing" << std::endl;
        return -1;
    }
    else 
    {
        std::cout << "Set GPU 0 for computing" << std::endl;
    }

    // get the supported priority on this deice
    int lowPriority = 0;
    int highPriority = 0;
    hipDeviceGetStreamPriorityRange(&lowPriority, &highPriority);
    std::cout << "lowPriority: " << lowPriority <<"\thighPriority: " << highPriority << std::endl;

    // set up data size of vectors
    int nElem = 1 << 24;

    // malloc host pinned memory
    
    float *pinned_A, *pinned_B, *h_C;
    size_t nBytes = nElem * sizeof(float);
    ErrorCheck(hipHostAlloc((void**)&pinned_A, nBytes, hipHostMallocDefault), __FILE__, __LINE__);
    ErrorCheck(hipHostAlloc((void**)&pinned_B, nBytes, hipHostMallocDefault), __FILE__, __LINE__);
    h_C = new float[nElem];

    if(pinned_A && pinned_B && h_C)
    {
        std::cout << "Allocate memory successfully!" <<std::endl;
    }
    else
    {
        std::cout << "Fail to allocate memory" <<std::endl;
        return -1;
    }

    // initialize data at host side
    initialData(pinned_A, nElem);
    initialData(pinned_B, nElem);

    // allocate GPU memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    // transfer data from host to device
    hipStream_t data_stream;
    hipStreamCreate(&data_stream);

    hipMemcpyAsync(d_A, pinned_A, nBytes, hipMemcpyHostToDevice, data_stream);
    hipEvent_t cp_evt_A;
    ErrorCheck(hipEventCreate(&cp_evt_A), __FILE__, __LINE__);
    
    hipMemcpyAsync(d_B, pinned_B, nBytes, hipMemcpyHostToDevice, data_stream);
    hipEvent_t cp_evt_B;
    ErrorCheck(hipEventCreate(&cp_evt_B), __FILE__, __LINE__);
    
    hipStreamSynchronize(data_stream);

    // calculate on GPU
    dim3 block (512);
    dim3 grid ((nElem + block.x - 1) / block.x, 1);

    hipStream_t kernelStream;
    hipStreamCreateWithPriority(&kernelStream, hipStreamDefault, highPriority);

    sumArraysOnGPU<<<grid, block, 0, kernelStream>>>(d_A, d_B, d_C, nElem);
    hipMemcpy(h_C, d_C, nBytes, hipMemcpyDeviceToHost);

    for(int i = 0; i < 50; ++i)
    {
        std::cout << "idx=" << i + 1 <<"\tmatrex_A: " << pinned_A[i] << "\tmatrix_B: " << pinned_B[i] << "\tresult=" << h_C[i] << std::endl;
    }


    hipHostFree (pinned_A);
    hipHostFree (pinned_B);
    delete[] h_C;
    hipFree (d_A);
    hipFree (d_B);
    hipFree (d_C);
    hipStreamDestroy(data_stream);

    hipEventDestroy(cp_evt_A);
    hipEventDestroy(cp_evt_B);

    hipDeviceReset();
    return 0;
}