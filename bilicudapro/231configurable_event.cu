#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstring>

#include "00common.h"

__global__ void infiniteKernel()
{
    while (true)
    {
        /* code */
    }
    
}

int main(int argc, char** argv)
{
    // get GPU device count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if (error != hipSuccess || nDeviceNumber == 0)
    {
        std::cout << "No CUDA campatable GPU found" << std::endl;
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if (error != hipSuccess || nDeviceNumber == 0)
    {
        std::cout << "Fail to set GPU 0 for computing" << std::endl;
        return -1;
    }
    else 
    {
        std::cout << "Set GPU 0 for computing" << std::endl;
    }

    
    int nElem = 16;

    // calculate on GPU
    dim3 block (nElem);
    dim3 grid (1);

    hipStream_t kernelStream;
    hipStreamCreate(&kernelStream);
    infiniteKernel<<<grid, block, 0, kernelStream>>>();

    hipEvent_t kernelEvent;
    ErrorCheck(hipEventCreateWithFlags(&kernelEvent, hipEventBlockingSync), __FILE__, __LINE__);
    ErrorCheck(hipEventRecord(kernelEvent, kernelStream), __FILE__, __LINE__);


    // wait for data copy to complete
    hipEventSynchronize(kernelEvent);
    std::cout << "Event kernelEvent is finished" << std::endl;


    hipStreamDestroy(kernelStream);
    hipEventDestroy(kernelEvent);

    hipDeviceReset();
    return 0;
}