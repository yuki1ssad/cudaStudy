#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstring>

#include "00common.h"

int main(int argc, char** argv)
{
    // get GPU device count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if (error != hipSuccess || nDeviceNumber == 0)
    {
        std::cout << "No CUDA campatable GPU found" << std::endl;
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if (error != hipSuccess || nDeviceNumber == 0)
    {
        std::cout << "Fail to set GPU 0 for computing" << std::endl;
        return -1;
    }
    else 
    {
        std::cout << "Set GPU 0 for computing" << std::endl;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    if (deviceProp.concurrentKernels)
    {
        std::cout << "concurrent kernel is supported on this GPU, begin to execute kernel_1" << std::endl;
    }
    else
    {
        std::cout << "concurrent kernel is Not supported on this GPU." << std::endl; 
    }
    
    hipDeviceReset();
    return 0;
}