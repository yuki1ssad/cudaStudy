#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstring>

#include "00common.h"

void initialData(float* ip, int size)
{
    time_t t;
    srand((unsigned) time(&t));
    // std::cout << "Matrix is: ";
    for (int i = 0; i < size; ++i)
    {
        ip[i] = static_cast<float>(rand() & 0xff) / 10.0;
        // std::cout << std::fixed << std::setprecision(2) << ip[i] << " ";
    }
    // std::cout <<std::endl;
    return;
}

void data_cp_callback(hipStream_t stream, hipError_t status, void* userData)
{
    printf("data copy callback is invoked, datasize: %d\n", *((int*)userData));
}

int main(int argc, char** argv)
{
    // get GPU device count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if (error != hipSuccess || nDeviceNumber == 0)
    {
        std::cout << "No CUDA campatable GPU found" << std::endl;
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if (error != hipSuccess || nDeviceNumber == 0)
    {
        std::cout << "Fail to set GPU 0 for computing" << std::endl;
        return -1;
    }
    else 
    {
        std::cout << "Set GPU 0 for computing" << std::endl;
    }

    int nElem = 1 << 12;

    size_t nBytes = nElem * sizeof(float);

    float* h_A;
    h_A = new float[nElem];

    initialData(h_A, nElem);

    float* d_A;
    hipMalloc((float**)&d_A, nBytes);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);

    // register call back
    hipStreamAddCallback(0, data_cp_callback, &nBytes, 0);

    delete[] h_A;
    hipFree (d_A);

    return 0;
}