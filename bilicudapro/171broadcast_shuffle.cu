#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#include "00common.h"

__global__ void shfl_broadcast(int *in, int *out, int const srcLane)
{
    int value = in[threadIdx.x];
    value = __shfl(value, srcLane, 32); // 将第 srcLane 条线程的值传递给其他线程的 value 变量
    out[threadIdx.x] = value;
}

int main(int argc, char** argv)
{
    // get GPU device count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if (error != hipSuccess || nDeviceNumber == 0)
    {
        std::cout << "No CUDA campatable GPU found" << std::endl;
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if (error != hipSuccess)
    {
        std::cout << "Fail to set GPU 0 for computing" << std::endl;
        return -1;
    }
    else 
    {
        std::cout << "Set GPU 0 for computing" << std::endl;
    }

    int nElem = 32;

    int *in = nullptr;
    int *out = nullptr;

    ErrorCheck(hipHostAlloc((void**)&in, sizeof(int) * nElem, hipHostMallocDefault), __FILE__, __LINE__);
    ErrorCheck(hipHostAlloc((void**)&out, sizeof(int) * nElem, hipHostMallocDefault), __FILE__, __LINE__);

    for (int i = 0; i < nElem; ++i)
    {
        in[i] = i;
    }

    // calculate on gpu
    dim3 block (nElem);
    dim3 grid (1);

    shfl_broadcast<<<grid, block>>>(in, out, 3);
    hipDeviceSynchronize();

    for (int i = 0; i < nElem; ++i)
    {
        std::cout << "out element is, id=" << i << "\tvalue=" << out[i] << std::endl;
    }

    hipHostFree(in);
    hipHostFree(out);
    hipDeviceReset();
    return 0;
}
