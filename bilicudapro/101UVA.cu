#include <hip/hip_runtime.h>
#include <iostream>

#include "00common.h"

int main(int argc, char** argv)
{
    // get GPU device count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if (error != hipSuccess || nDeviceNumber == 0)
    {
        std::cout << "No CUDA campatable GPU found" << std::endl;
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if (error != hipSuccess)
    {
        std::cout << "Fail to set GPU 0 for computing" << std::endl;
        return -1;
    }
    else 
    {
        std::cout << "Set GPU 0 for computing" << std::endl;
    }

    float *d_mem = nullptr;
    ErrorCheck(hipMalloc((void**)&d_mem, sizeof(float)), __FILE__, __LINE__);

    hipPointerAttribute_t pt_Attribute;
    ErrorCheck(hipPointerGetAttributes(&pt_Attribute, d_mem), __FILE__, __LINE__);
    std::cout << "pointer Attribute:device=" << pt_Attribute.device << "\tdevicePointer=" << pt_Attribute.devicePointer << "\ttype=" <<pt_Attribute.type << std::endl;

    hipFree (d_mem);
    hipDeviceReset();
    return 0;
}