#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstring>

#include "00common.h"


__global__ void standard_kernel(float a, float *out, int iters)
{
    // int i;
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

    if (tid == 0)
    {
        float tmp;
        for (int i = 0; i < iters; ++i)
        {
            tmp = powf(a, 2.0f);
        }
        *out = tmp;
    }
}

__global__ void intrinsic_kernel(float a, float *out, int iters)
{
    // int i;
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

    if (tid == 0)
    {
        float tmp;
        for (int i = 0; i < iters; ++i)
        {
            tmp = __powf(a, 2.0f);
        }
        *out = tmp;
    }
}

int main(int argc, char** argv)
{
    // get GPU device count
    int nDeviceNumber = 0;
    hipError_t error = ErrorCheck(hipGetDeviceCount(&nDeviceNumber), __FILE__, __LINE__);
    if (error != hipSuccess || nDeviceNumber == 0)
    {
        std::cout << "No CUDA campatable GPU found" << std::endl;
        return -1;
    }

    // set up device
    int dev = 0;
    error = ErrorCheck(hipSetDevice(dev), __FILE__, __LINE__);
    if (error != hipSuccess || nDeviceNumber == 0)
    {
        std::cout << "Fail to set GPU 0 for computing" << std::endl;
        return -1;
    }
    else 
    {
        std::cout << "Set GPU 0 for computing" << std::endl;
    }

    int iters = 1000;
    float input_value = 0.0f;

    if (argc == 3)
    {
        iters = atoi(argv[2]);
        input_value = (float)atof(argv[1]);
    }

    std::cout << "iteration is " << iters << "\tinput value is " << input_value << std::endl;

    // int i;
    int runs = 30;


    float *d_standard_out, h_standart_out;
    hipMalloc((void**)&d_standard_out, sizeof(float));
    
    float *d_intrinsic_out, h_intrinsic_out;
    hipMalloc((void**)&d_intrinsic_out, sizeof(float));

    double mean_intrinsic_time = 0.0, mean_standard_time = 0.0;

    for (int i = 0; i < runs; ++i)
    {
        double start_standard = GetCPUSecond();
        standard_kernel<<<1, 32>>>(input_value, d_standard_out, iters);
        hipDeviceSynchronize();
        mean_standard_time += GetCPUSecond() - start_standard;

        double start_intrinsic = GetCPUSecond();
        intrinsic_kernel<<<1, 32>>>(input_value, d_intrinsic_out, iters);
        hipDeviceSynchronize();
        mean_intrinsic_time += GetCPUSecond() - start_intrinsic;
    }

    hipMemcpy(&h_standart_out, d_standard_out, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_intrinsic_out, d_intrinsic_out, sizeof(float), hipMemcpyDeviceToHost);

    float host_value = powf(input_value, 2.0f);

    printf("Host calculated\t\t\t%f\n", host_value);
    printf("Standard Device calculated\t%f\n", h_standart_out);
    printf("Intrinsic Device calculated\t%f\n", h_intrinsic_out);
    printf("Host equals Standard?\t\t%s diff=%e\n", host_value == h_standart_out ? "YES" : "NO", fabs(host_value - h_standart_out));
    printf("Host equals Intrinsic?\t\t%s diff=%e\n", host_value == h_intrinsic_out ? "YES" : "NO", fabs(host_value - h_intrinsic_out));
    printf("Standard equals Intrinsic?\t\t%s diff=%e\n", h_standart_out == h_intrinsic_out ? "YES" : "NO", fabs(h_standart_out - h_intrinsic_out));
    printf("\n");
    printf("Mean execution time for standard function powf: %f s\n", mean_standard_time);
    printf("Mean execution time for intrinsic function __powf: %f s\n", mean_intrinsic_time);

    hipFree (d_standard_out);
    hipFree (d_intrinsic_out);
    hipDeviceReset();
    return 0;
}

/*
    nvcc --ptx 291intrinsic_standard_func.cu -o is.ptx
*/